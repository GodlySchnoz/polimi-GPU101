#include <cstdlib>
#include <fstream>
#include <iostream>
#include <vector>
#include <climits>
#include <hip/hip_runtime.h>

#define MAX_FRONTIER_SIZE 128

#define CHECK(call)                                                                     \
{                                                                                       \
    const hipError_t err = call;                                                       \
    if (err != hipSuccess) {                                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}

#define CHECK_KERNELCALL()                                                              \
{                                                                                       \
    const hipError_t err = hipGetLastError();                                         \
    if (err != hipSuccess) {                                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}

__global__ void bfs_ker(int *Va, int *Ea, char *Fa, char *Xa, int *Ca, int num_vertex) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_vertex || !Fa[tid]) return; // Escape condition: Out of bounds or inactive frontier

    Fa[tid] = 0; // Mark this node as processed
    Xa[tid] = 1; // Mark this node as explored

    for (int i = Va[tid]; i < Va[tid + 1]; ++i) {
        int neighbor = Ea[i];
        if (!Xa[neighbor]) { // If neighbor not explored
            Fa[neighbor] = 1;
            atomicMin(&Ca[neighbor], Ca[tid] + 1);
        }
    }
}

void bfs_cuda(std::vector<int> &Va, std::vector<int> &Ea, int source, int num_vertex) {
    int *Va_d, *Ea_d, *Ca_d;
    char *Fa_d, *Xa_d;

    // Host-side initialization
    std::vector<char> Fa(num_vertex, 0);
    std::vector<char> Xa(num_vertex, 0);
    std::vector<int> Ca(num_vertex, INT_MAX);

    Fa[source] = 1;
    Ca[source] = 0;

    // Device memory allocation
    CHECK(hipMalloc((void**)&Va_d, Va.size() * sizeof(int)));
    CHECK(hipMalloc((void**)&Ea_d, Ea.size() * sizeof(int)));
    CHECK(hipMalloc((void**)&Ca_d, num_vertex * sizeof(int)));
    CHECK(hipMemcpy(Ca_d, Ca.data(), num_vertex * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMalloc((void**)&Fa_d, num_vertex * sizeof(char)));
    CHECK(hipMemset(Fa_d, 0, num_vertex * sizeof(char)));
    CHECK(hipMalloc((void**)&Xa_d, num_vertex * sizeof(char)));
    CHECK(hipMemset(Xa_d, 0, num_vertex * sizeof(char)));

    // Copy data to device
    CHECK(hipMemcpy(Va_d, Va.data(), Va.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(Ea_d, Ea.data(), Ea.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(Ca_d, Ca.data(), num_vertex * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(Fa_d, Fa.data(), num_vertex * sizeof(char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(Xa_d, Xa.data(), num_vertex * sizeof(char), hipMemcpyHostToDevice));

    // Kernel launch configuration
    int tpb = 256; // Threads per block
    int bpg = (num_vertex + tpb - 1) / tpb; // Blocks per grid

    bool continue_bfs;
    do {
        continue_bfs = false;
        bfs_ker<<<bpg, tpb>>>(Va_d, Ea_d, Fa_d, Xa_d, Ca_d, num_vertex);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());

        // Copy the Fa array back to the host to check if more processing is needed
        CHECK(hipMemcpy(Fa.data(), Fa_d, num_vertex * sizeof(char), hipMemcpyDeviceToHost));

        for (int i = 0; i < num_vertex; ++i) {
            if (Fa[i]) {
                continue_bfs = true;
                break;
            }
        }
    } while (continue_bfs);

    // Copy final distances back to the host
    CHECK(hipMemcpy(Ca.data(), Ca_d, num_vertex * sizeof(int), hipMemcpyDeviceToHost));

    // Print results
    for (int i = 0; i < num_vertex; ++i) {
        std::cout << "Vertex " << i + 1 << " Distance: " << Ca[i] << std::endl;
    }

    // Free device memory
    hipFree(Va_d);
    hipFree(Ea_d);
    hipFree(Ca_d);
    hipFree(Fa_d);
    hipFree(Xa_d);
}

void read_matrix(std::vector<int> &Va, std::vector<int> &Ea, const std::string &filename, int &num_vertex, int &num_cols, int &num_edges) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("File cannot be opened!");
    }

    // Get matrix dimensions
    file >> num_vertex >> num_cols >> num_edges;
    if (num_vertex <= 0 || num_cols <= 0 || num_edges <= 0) {
        throw std::runtime_error("Invalid matrix dimensions in file!");
    }

    Va.resize(num_vertex + 1, 0);
    Ea.resize(num_edges);

    // Count row occurrences
    std::vector<int> row_counts(num_vertex, 0);
    int row, col;
    float val;

    while (file >> row >> col >> val) {
        row--; // Adjust to 0-based indexing
        row_counts[row]++;
    }

    // Construct Va
    int edge_index = 0;
    for (int i = 0; i < num_vertex; ++i) {
        Va[i] = edge_index;
        edge_index += row_counts[i];
    }
    Va[num_vertex] = num_edges;

    file.clear();
    file.seekg(0);

    file >> num_vertex >> num_cols >> num_edges; // Skip header line

    // Fill Ea
    std::vector<int> row_offsets(num_vertex, 0);
    while (file >> row >> col >> val) {
        row--; col--;
        int position = Va[row] + row_offsets[row];
        Ea[position] = col;
        row_offsets[row]++;
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: ./exec matrix_file source\n";
        return EXIT_FAILURE;
    }

    std::vector<int> Va, Ea;
    int num_vertex, num_cols, num_edges;

    const std::string filename(argv[1]);
    int source = atoi(argv[2]) - 1;

    try {
        read_matrix(Va, Ea, filename, num_vertex, num_cols, num_edges);
        bfs_cuda(Va, Ea, source, num_vertex);
    } catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
