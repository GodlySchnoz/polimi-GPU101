#include <cstdlib>
#include <fstream>
#include <iostream>
#include <vector>
#include <climits>
#include <hip/hip_runtime.h>

#define MAX_FRONTIER_SIZE 128

#define CHECK(call)                                                                     \
{                                                                                       \
    const hipError_t err = call;                                                       \
    if (err != hipSuccess) {                                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}

#define CHECK_KERNELCALL()                                                              \
{                                                                                       \
    const hipError_t err = hipGetLastError();                                         \
    if (err != hipSuccess) {                                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}

__global__ void bfs_ker(int *Va, int *Ea, char *Fa, char *Xa, int *Ca, int num_vertex) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_vertex || !Fa[tid]) return; // Escape condition: Out of bounds or inactive frontier

    Fa[tid] = 0; // Mark this node as processed
    Xa[tid] = 1; // Mark this node as explored

    for (int i = Va[tid]; i < Va[tid + 1]; ++i) {
        int neighbor = Ea[i];
        if (!Xa[neighbor]) { // If neighbor not explored
            Fa[neighbor] = 1;
            atomicMin(&Ca[neighbor], Ca[tid] + 1);
        }
    }
}

void bfs_cuda(std::vector<int> &Va, std::vector<int> &Ea, int source, int num_vertex) {
    int *Va_d, *Ea_d, *Ca_d;
    char *Fa_d, *Xa_d;

    // Initialize host data
    std::vector<char> Fa(num_vertex, 0);
    std::vector<char> Xa(num_vertex, 0);
    std::vector<int> Ca(num_vertex, INT_MAX);
    Fa[source] = 1;
    Ca[source] = 0;

    // Allocate device memory
    hipMalloc((void**)&Va_d, Va.size() * sizeof(int));
    hipMalloc((void**)&Ea_d, Ea.size() * sizeof(int));
    hipMalloc((void**)&Ca_d, num_vertex * sizeof(int));
    hipMemcpy(Ca_d, Ca.data(), num_vertex * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&Fa_d, num_vertex * sizeof(char));
    hipMemset(Fa_d, 0, num_vertex * sizeof(char));
    hipMalloc((void**)&Xa_d, num_vertex * sizeof(char));
    hipMemset(Xa_d, 0, num_vertex * sizeof(char));

    hipMemcpy(Va_d, Va.data(), Va.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Ea_d, Ea.data(), Ea.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Fa_d, Fa.data(), num_vertex * sizeof(char), hipMemcpyHostToDevice);

    int tpb = 256;
    int bpg = (num_vertex + tpb - 1) / tpb;

    // Create CUDA streams
    hipStream_t compute_stream, transfer_stream;
    hipStreamCreate(&compute_stream);
    hipStreamCreate(&transfer_stream);

    bool continue_bfs;

    do {
        continue_bfs = false;
        // use the compute_stream for the kernel call (HOST -> DEVICE)
        bfs_ker<<<bpg, tpb, 0, compute_stream>>>(Va_d, Ea_d, Fa_d, Xa_d, Ca_d, num_vertex);
        // use the transfer_stream for the memcpy/synch calls (DEVICE -> HOST)
        hipMemcpyAsync(Fa.data(), Fa_d, num_vertex * sizeof(char), hipMemcpyDeviceToHost, transfer_stream);

        hipStreamSynchronize(transfer_stream);

        for (int i = 0; i < num_vertex; ++i) {
            if (Fa[i]) {
                continue_bfs = true;
                break;
            }
        }
    } while (continue_bfs);

    // Copy final distances back to the host
    hipMemcpy(Ca.data(), Ca_d, num_vertex * sizeof(int), hipMemcpyDeviceToHost);

    // // Print distances, used for testing, suppressed for final submission
    // for (int i = 0; i < num_vertex; ++i) {
    //     std::cout << "Vertex " << i + 1 << " Distance: " << Ca[i] << std::endl;
    // }

    // Clean up
    hipFree(Va_d);
    hipFree(Ea_d);
    hipFree(Ca_d);
    hipFree(Fa_d);
    hipFree(Xa_d);

    hipStreamDestroy(compute_stream);
    hipStreamDestroy(transfer_stream);
}

void read_matrix(std::vector<int> &Va, std::vector<int> &Ea, const std::string &filename, int &num_vertex, int &num_cols, int &num_edges) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("File cannot be opened!");
    }

    // Get matrix dimensions
    file >> num_vertex >> num_cols >> num_edges;
    if (num_vertex <= 0 || num_cols <= 0 || num_edges <= 0) {
        throw std::runtime_error("Invalid matrix dimensions in file!");
    }

    Va.resize(num_vertex + 1, 0);
    Ea.resize(num_edges);

    // Count row occurrences
    std::vector<int> row_counts(num_vertex, 0);
    int row, col;
    float val;

    while (file >> row >> col >> val) {
        row--; // Adjust to 0-based indexing
        row_counts[row]++;
    }

    // Construct Va
    int edge_index = 0;
    for (int i = 0; i < num_vertex; ++i) {
        Va[i] = edge_index;
        edge_index += row_counts[i];
    }
    Va[num_vertex] = num_edges;

    file.clear();
    file.seekg(0);

    file >> num_vertex >> num_cols >> num_edges; // Skip header line

    // Fill Ea
    std::vector<int> row_offsets(num_vertex, 0);
    while (file >> row >> col >> val) {
        row--; col--;
        int position = Va[row] + row_offsets[row];
        Ea[position] = col;
        row_offsets[row]++;
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: ./exec matrix_file source\n";
        return EXIT_FAILURE;
    }

    std::vector<int> Va, Ea;
    int num_vertex, num_cols, num_edges;

    const std::string filename(argv[1]);
    int source = atoi(argv[2]) - 1;

    try {
        read_matrix(Va, Ea, filename, num_vertex, num_cols, num_edges);
        bfs_cuda(Va, Ea, source, num_vertex);
    } catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
